/* --------------------------------- Header --------------------------------- */


/**
 * @file    cudaerr.cu
 * @brief   Contains CUDA error-handling functions
 */


/* -------------------------------- Includes -------------------------------- */


# include <iostream>

# include "cudaerr.cuh"


/* -------------------------------- Namespace ------------------------------- */


using namespace std;


/* -------------------------------- Functions ------------------------------- */


/**
 * Handles a CUDA error
 *
 * @param err   The CUDA error code
 * @param file  The file the error occurred in
 * @param line  The line the error occurred on
 */
void HandleCUDAError( hipError_t err, const char *file, int line )
{
    if ( err != hipSuccess )
    {
        cout << "CUDA ERROR: " << hipGetErrorString( err );
        cout << " in " << file << " at line " << line << endl;
        exit(1);
    }
}


/* -------------------------------------------------------------------------- */