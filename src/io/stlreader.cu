/* --------------------------------- Header --------------------------------- */


/**
 * @file    stlreader.cpp
 * @brief   STL file reader
 */


/* -------------------------------- Includes -------------------------------- */


# include <sstream>

# include "stlreader.h"


/* ----------------------- Constructors / Destructors ----------------------- */


/**
 * @brief   Creates an STL reader
 *
 * @param   void
 *
 * @return  The created STL reader
 */
STLReader::STLReader() = default;


/**
 * @brief   Creates an STL reader opens the specified STL file
 *
 * @param   *filePath   The path of the STL file to open
 *
 * @return  The created STL reader
 */
STLReader::STLReader( const char *filePath )
{
    open( ( std::string& ) filePath );
}


/**
 * @brief   Creates an STL reader opens the specified STL file
 *
 * @param   &filePath   The path of the STL file to open
 *
 * @return  The created STL reader
 */
STLReader::STLReader( const std::string &filePath )
{
    open( filePath );
}


/**
 * @brief   Creates an STL reader from an existing STL reader
 *
 * @param   &stlReader  The STL reader to create from
 *
 * @return  The created STL reader
 */
STLReader::STLReader( const STLReader &stlReader ) = default;


/**
 * @brief   STL reader destructor
 *
 * @param   void
 *
 * @return  void
 */
STLReader::~STLReader()
{
    // close if a file is open
    if ( isOpen ) close();

    // free file stream
    delete in;
}


/* ---------------------------- Public Functions ---------------------------- */


/**
 * @brief   Opens an STL file in this STL reader
 *
 * @param   &filePath   The path of the STL file to open
 *
 * @return  void
 */
void STLReader::open( const std::string &filePath )
{
    // close file if already open
    if ( isOpen ) close();

    // open an input file stream
    in->open( filePath.c_str() );

    // throw error if file fails to open
    if ( !in->is_open() )
    {
        throw STLReaderException( "File does not exist." );
    }

    // set open flag
    isOpen = true;
}


/**
 * @brief   Closes this STL reader
 *
 * @param   void
 *
 * @return  void
 */
void STLReader::close()
{
    // close input file stream
    in->close();

    // clear open flag
    isOpen = false;
}


/**
 * @brief   Gets the number of facets in the currently opened STL file
 *
 * @param   void
 *
 * @return  The number of facets in the currently opened STL file
 */
unsigned int STLReader::getFacetCount()
{
    // throw an exception if not open
    if ( !isOpen )
    {
        throw STLReaderException( "No open file." );
    }

    // initialize facet count
    unsigned int facetCount = 0;

    // count facets
    while ( *in )
    {
        std::string line;
        getline( *in, line );

        std::string token;
        std::stringstream ss = std::stringstream( line );
        ss >> token;

        if ( ( token == "facet" ) ) facetCount++;
    }

    // seek back to the beginning of the input stream
    in->clear();
    in->seekg( 0 );

    return facetCount;
}


Triangle STLReader::readFacet( unsigned int index )
{
    // throw an exception if not open
    if ( !isOpen )
    {
        throw STLReaderException( "No open file." );
    }

    // iterate to facet at index
    unsigned int facetCount = 0;

    while ( *in && ( facetCount <= index ) )
    {
        std::string line;
        getline( *in, line );

        std::string token;
        std::stringstream ss = std::stringstream( line );
        ss >> token;

        if ( ( token == "facet" ) ) facetCount++;
    }

    // throw end-of-file exception
    if ( !*in )
    {
        throw STLReaderException( "Facet index out of range." );
    }

    // advance the stream 1 position
    std::string garbage;
    getline( *in, garbage );

    // create array of 3D points to store vertices
    Point3D verts[3];

    // read the vertices
    for ( unsigned int i = 0; i < 3; i++ )
    {
        // get string stream for current vertex string
        std::string vertexString;
        getline( *in, vertexString );

        std::string token;
        std::stringstream ss = std::stringstream( vertexString );
        ss >> token;

        // throw an exception if vertex string is invalid
        if ( token != "vertex" )
        {
            throw STLReaderException( "Invalid or corrupt file format." );
        }

        // read vertex coords
        double x;
        double y;
        double z;

        ss >> x;
        ss >> y;
        ss >> z;

        // create vertex
        verts[i] = Point3D( x, y, z );
    }

    // seek back to the beginning of the input stream
    in->clear();
    in->seekg( 0 );

    // create a facet from the verts
    Triangle facet = Triangle( verts[0], verts[1], verts[2] );

    return facet;
}


ShapeContainer STLReader::readFacets()
{
    // throw an exception if not open
    if ( !isOpen )
    {
        throw STLReaderException( "No open file." );
    }

    // create shape container
    ShapeContainer sc = ShapeContainer();

    // advance to next facet line
    while ( *in )
    {
        // get line
        std::string line;
        getline( *in, line );

        std::string token;
        std::stringstream ss = std::stringstream( line );
        ss >> token;

        // process if facet
        if (( token == "facet" ))
        {
            // advance the stream 1 position
            getline( *in, line );

            // create array of 3D points to store vertices
            Point3D verts[3];

            // read the vertices
            for ( unsigned int i = 0; i < 3; i++ )
            {
                // get string stream for current vertex string
                getline( *in, line );

                ss = std::stringstream( line );
                ss >> token;

                // throw an exception if vertex string is invalid
                if ( token != "vertex" )
                {
                    throw STLReaderException( "Invalid or corrupt file format." );
                }

                // read vertex coords
                double x;
                double y;
                double z;

                ss >> x;
                ss >> y;
                ss >> z;

                // create vertex
                verts[i] = Point3D( x, y, z );
            }

            // create a facet from the verts
            Triangle facet = Triangle( verts[0], verts[1], verts[2] );

            // add the facet to the shape container
            sc.add( facet );
        }
    }

    return sc;
}


/* -------------------------------------------------------------------------- */