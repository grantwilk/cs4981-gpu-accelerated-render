#include "hip/hip_runtime.h"
/* --------------------------------- Header --------------------------------- */


/**
 * @file    point3D.cpp
 * @brief   3D point class
 */


/* -------------------------------- Includes -------------------------------- */


# include <algorithm>
# include <cmath>

# include "point3d.h"


/* ----------------------- Constructors / Destructors ----------------------- */


/**
 * @brief   Creates a 3D point at coordinates (0, 0)
 *
 * @param   x   The x-coordinate
 * @param   y   The y-coordinate
 *
 * @return  The created point
 */
Point3D::Point3D():
    Point3D( 0, 0, 0 )
{}


/**
 * @brief   Creates a 3D point at the specified coordinates
 *
 * @param   x   The x-coordinate
 * @param   y   The y-coordinate
 * @param   z   The z-coordinate
 *
 * @return  The created point
 */
Point3D::Point3D( float x, float y, float z ):
    Vector3<float>( x, y, z )
{}


/**
 * @brief   Creates a 3D point from a 3D vector of floats
 *
 * @param   &v  The vector to create the 3D point from
 *
 * @return  The created point
 */
Point3D::Point3D( const Vector3<float> &v ):
    Point3D( v[0], v[1], v[2] )
{}


/**
 * @brief   Creates a 3D point from a 2x1 matrix of floats
 *
 * @param   &m  The matrix to create the 3D point from
 *
 * @return  The created point
 */
Point3D::Point3D( const Matrix<float> &m ):
    Point3D( m[0][0], m[1][0], m[2][0] )
{
    if ( ( m.getRows() != 3 ) || ( m.getColumns() != 1 ) )
    {
        throw MatrixException( "Point3D construction matrix size mismatch" );
    }
}


/**
 * @brief   Creates a 3D point from an existing 3D point
 *
 * @param   point   The 3D point to create from
 *
 * @return  The created point
 */
Point3D::Point3D( const Point3D &p ):
    Point3D( p.getX(), p.getY(), p.getZ() )
{}


Point3D::~Point3D() = default;


/* -------------------------- Overloaded Operators -------------------------- */


/**
 * @brief   Assigns a 3D point to this 3D point
 *
 * @param   &p  The 3D point to assign
 *
 * @return  A reference to this 3D point
 */
Point3D &Point3D::operator=( const Point3D &p )
{
    setX( p.getX() );
    setY( p.getY() );
    setZ( p.getZ() );
    return *this;
}


/**
 * @brief   Adds a 3D point to this 3D point and assigns the sum to
 *          this 3D point
 *
 * @param   &p  The 3D point to add
 *
 * @return  A reference to this 3D point
 */
Point3D &Point3D::operator+=( const Point3D &p )
{
    return *this = *this + p;
}


/**
 * @brief   Subtracts a 3D point from this 3D point and assigns the
 *          difference to this 3D point
 *
 * @param   &p  The 3D point to subtract
 *
 * @return  A reference to this 3D point
 */
Point3D &Point3D::operator-=( const Point3D &p )
{
    return *this = *this - p;
}


/**
 * @brief   Adds a 3D point to this 3D point
 *
 * @param   &p  The 3D point to add
 *
 * @return  The sum of the 3D points
 */
Point3D Point3D::operator+( const Point3D &p ) const
{
    return Point3D( getX() + p.getX(), getY() + p.getY(), getZ() + p.getZ() );
}


/**
 * @brief   Subtracts a 3D point from this 3D point
 *
 * @param   &p  The 3D point to subtract
 *
 * @return  The difference of the 3D points
 */
Point3D Point3D::operator-( const Point3D &p ) const
{
    return Point3D( getX() - p.getX(), getY() - p.getY(), getZ() - p.getZ() );
}


/**
 * @brief   Determines if the magnitude of this 3D point is less than the
 *          magnitude of another 3D point
 *
 * @param   &p  The 3D point to compare against
 *
 * @return  True if the magnitude of this 3D point is less than the magnitude
 *          of the other point, false otherwise
 */
bool Point3D::operator<( const Point3D &p ) const
{
    return magnitude() < p.magnitude();
}


/**
 * @brief   Determines if the magnitude of this 3D point is less than or equal
 *          to the magnitude of another 3D point
 *
 * @param   &p  The 3D point to compare against
 *
 * @return  True if the magnitude of this 3D point is less than or equal to
 *          the magnitude of the other point, false otherwise
 */
bool Point3D::operator<=( const Point3D &p ) const
{
    return magnitude() <= p.magnitude();
}


/**
 * @brief   Determines if the magnitude of this 3D point is greater than the
 *          magnitude of another 3D point
 *
 * @param   &p  The 3D point to compare against
 *
 * @return  True if the magnitude of this 3D point is greater than the magnitude
 *          of the other point, false otherwise
 */
bool Point3D::operator>( const Point3D &p ) const
{
    return magnitude() > p.magnitude();
}


/**
 * @brief   Determines if the magnitude of this 3D point is greater than or
 *          equal to the magnitude of another 3D point
 *
 * @param   &p  The 3D point to compare against
 *
 * @return  True if the magnitude of this 3D point is greater than or equal to
 *          the magnitude of the other point, false otherwise
 */
bool Point3D::operator>=( const Point3D &p ) const
{
    return magnitude() >= p.magnitude();
}


/**
 * @brief   Converts a 3D point to a string and writes it to an output stream
 *
 * @param   &os     The output stream to write to
 * @param   &p      The 3D point to convert
 *
 * @return  The output stream
 */
std::ostream &operator<<( std::ostream &os, const Point3D &p )
{
    p.out( os );
    return os;
}


/* ---------------------------- Public Functions ---------------------------- */



/**
 * @brief   Creates a dynamically allocated clone of this 3D point
 *
 * @param   void
 *
 * @return  A pointer to cloned 3D point
 */
Point3D *Point3D::clone() const
{
    return new Point3D( getX(), getY(), getZ() );
}


/**
 * @brief   Calculates a transformed copy of this 3D point
 *
 * @param   &m  The 2D transformation matrix
 *
 * @return  A transformed copy of this 3D point
 */
Point3D Point3D::transform( const Matrix<float> &m ) const
{
    if ( ( m.getRows() != 4 ) || ( m.getColumns() != 4 ) )
    {
        throw MatrixException( "Point3D transformation matrix size mismatch." );
    }

    Matrix<float> transformVector = m * Vector4<float>( getX(), getY(), getZ(), 1 );

    return Point3D( transformVector[0][0], transformVector[1][0], transformVector[2][0] );
}


/**
 * @brief   Gets the magnitude of the vector between the origin and this
 *          3D point
 *
 * @param   void
 *
 * @return  The magnitude of the vector between the origin and this 3D point
 */
float Point3D::magnitude() const
{
    return sqrt( pow( getX(), 2 ) + pow( getY(), 2 ) + pow( getZ(), 2 ) );
}


/**
 * @brief   Gets the angle of the vector between this 3D point and another
 *          3D point in radians
 *
 * @param   void
 *
 * @return  The angle of the vector between this 3D point and another 2D
 *          point in radians
 */
float Point3D::angle( const Point3D &p ) const
{
    float dot = this->dot( p );
    float magSquared = pow( this->magnitude(), 2 );
    float pMagSquared = pow( p.magnitude(), 2 );
    return acos( dot / sqrt( magSquared * pMagSquared ) );
}


/**
 * @brief   Converts this 3D point to a string and writes it to an output
 *          stream
 *
 * @param   &os     The output stream to write to
 *
 * @return  The output stream
 */
std::ostream &Point3D::out( std::ostream &os ) const
{
    os << "Point3D( " << getX() << " " << getY() << " " << getZ() << " )";
    return os;
}


/**
 * @brief   Performs a deep copy of a vector of 3D point pointers
 *
 * @param   &v      The vector to copy
 *
 * @return  The copied vector
 */
std::vector<Point3D*> Point3D::vectorDeepCopy( const std::vector<Point3D*> &v )
{
    std::vector<Point3D*> clone = std::vector<Point3D*>();

    std::for_each( v.begin(), v.end(), [&clone]( Point3D *p )
       {
           Point3D *pClone = p->clone();
           clone.push_back( pClone );
       }
    );

    return clone;
}


/**
 * @brief   Performs a deep delete of a vector of 3D point pointers
 *
 * @param   &v      The vector to delete
 *
 * @return  void
 */
void Point3D::vectorDeepDelete( const std::vector<Point3D*> &v )
{
    std::for_each( v.begin(), v.end(), []( Point3D *p )
       {
           delete p;
       }
    );
}


/**
 * @brief   Performs a deep copy a 3D vector of 3D point pointers
 *
 * @param   &v      The vector to copy
 *
 * @return  The copied vector
 */
Vector2<Point3D*> Point3D::vector2DeepCopy( const Vector2<Point3D*> &v )
{
    return Vector2<Point3D*>( v[0]->clone(), v[1]->clone() );
}


/**
 * @brief   Performs a deep deletion of a 3D vector of 3D point pointers
 *
 * @param   &v      The vector to delete
 *
 * @return  void
 */
void Point3D::vector2DeepDelete( const Vector2<Point3D*> &v )
{
    delete v.getX();
    delete v.getY();
}


/**
 * @brief   Performs a deep copy a 3D vector of 3D point pointers
 *
 * @param   &v      The vector to copy
 *
 * @return  The copied vector
 */
Vector3<Point3D*> Point3D::vector3DeepCopy( const Vector3<Point3D*> &v )
{
    return Vector3<Point3D*>( v[0]->clone(), v[1]->clone(), v[2]->clone() );
}


/**
 * @brief   Performs a deep deletion of a 3D vector of 3D point pointers
 *
 * @param   &v      The vector to delete
 *
 * @return  void
 */
void Point3D::vector3DeepDelete( const Vector3<Point3D*> &v )
{
    delete v.getX();
    delete v.getY();
    delete v.getZ();
}


/**
 * @brief   Performs a deep copy a 4D vector of 3D point pointers
 *
 * @param   &v      The vector to copy
 *
 * @return  The copied vector
 */
Vector4<Point3D*> Point3D::vector4DeepCopy( const Vector4<Point3D*> &v )
{
    return Vector4<Point3D*>( v[0]->clone(), v[1]->clone(), v[2]->clone(), v[3]->clone() );
}


/**
 * @brief   Performs a deep deletion of a 3D vector of 3D point pointers
 *
 * @param   &v      The vector to delete
 *
 * @return  void
 */
void Point3D::vector4DeepDelete( const Vector4<Point3D*> &v )
{
    delete v.getX();
    delete v.getY();
    delete v.getZ();
    delete v.getW();
}


/* -------------------------------------------------------------------------- */