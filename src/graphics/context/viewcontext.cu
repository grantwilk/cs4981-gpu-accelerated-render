#include "hip/hip_runtime.h"
/* --------------------------------- Header --------------------------------- */


/**
 * @file    viewcontext.cpp
 * @brief   View context for view transformations
 */


/* -------------------------------- Includes -------------------------------- */


# include <cmath>

# include "cudaerr.cuh"
# include "viewcontext.h"

/* ---------------------------- Static Variables ---------------------------- */


float * ViewContext::d_viewTransform = nullptr;
Matrix<float> ViewContext::transform = Matrix<float>(4, 4);
Matrix<float> ViewContext::invTransform = Matrix<float>(4, 4);


/* ----------------------- Constructors / Destructors ----------------------- */


/**
 * @brief   Creates a view context object
 *
 * @param   *gc     The graphics context to pair with this view context
 *
 * @return  The created view context
 */
ViewContext::ViewContext( GraphicsContext *gc ) :
    gc( gc )
{
    HANDLE_CUDA_ERROR(
        hipMalloc(&d_viewTransform, 4 * 4 * sizeof(float))
    );

    update();
}


/**
 * @brief   View context destructor
 *
 * @param   void
 *
 * @return  void
 */
ViewContext::~ViewContext() = default;


/* ---------------------------- Public Functions ---------------------------- */


/**
 * @brief   Transforms a 3D model point to a 3D device point through the view
 *          context's view transformation matrix
 *
 * @param   &p  The 3D model point to transform
 *
 * @return  The transformed 3D device point
 */
Point3D ViewContext::modelToDevice( const Point3D &p )
{
    return p.transform( transform );
}


/**
 * @brief   Transforms a 3D device point to a 3D model point through the view
 *          context's view transformation matrix
 *
 * @param   &p  The 3D device point to transform
 *
 * @return  The transformed 3D model point
 */
Point3D ViewContext::deviceToModel( const Point3D &p )
{
    return p.transform( invTransform );
}


/**
 * @brief   Gets a unit vector pointing in the direction of the view plane's look direction
 *
 * @param   void
 *
 * @return  A unit vector pointing in direction of the view plane's look direction
 */
Point3D ViewContext::getLookVector()
{
    Point3D lookVector = Point3D( 0, 0, 1 );
    return lookVector.transform( genInvViewRotationMatrix() );
}


/**
 * @brief   Applies a view translation to the transformation matrix
 *
 * @param   x   The magnitude of the translation along the +x axis in model units
 * @param   y   The magnitude of the translation along the +y axis in model units
 * @param   z   The magnitude of the translation along the +z axis in model units
 *
 * @return  void
 */
void ViewContext::translate( float x, float y, float z )
{
    setTranslation(
        viewTranslationX + x,
        viewTranslationY + y,
        viewTranslationZ + z
    );
}


/**
 * @brief   Applies a view rotation to the transformation matrix
 *
 * @param   x   The magnitude of the rotation along the +x axis in radians
 * @param   y   The magnitude of the rotation along the +y axis in radians
 *
 * @return  void
 */
void ViewContext::rotate( float x, float y )
{
    setRotation(
        viewRotationX + x,
        viewRotationY + y
    );
}


/**
 * @brief   Applies a view scale to the transformation matrix
 *
 * @param   x   The magnitude of the scale along the +x axis
 * @param   y   The magnitude of the scale along the +y axis
 * @param   z   The magnitude of the scale along the +y axis
 *
 * @return  void
 */
void ViewContext::scale( float x, float y, float z )
{
    setScale(
        viewScaleX * x,
        viewScaleY * y,
        viewScaleZ * z
    );
}


/**
 * @brief   Translates the view along the x and y axis as pertains to the view
 *          plane of the view transform
 *
 * @param   x   The magnitude of the translation on the view plane's +x axis
 * @param   y   The magnitude of the translation on the view plane's +y axis
 *
 * @return  void
 */
void ViewContext::pan( float x, float y )
{
    panX( x );
    panY( y );
}


/**
 * @brief   Sets the translation of the view context's transformation matrix
 *
 * @param   x   The view transform translation along the +x axis in model units
 * @param   y   The view transform translation along the +y axis in model units
 * @param   z   The view transform translation along the +z axis in model units
 *
 * @return  void
 */
void ViewContext::setTranslation( float x, float y, float z )
{
    viewTranslationX = x;
    viewTranslationY = y;
    viewTranslationZ = z;
    update();
}


/**
 * @brief   Sets the rotation of the view context's transformation matrix
 *
 * @param   x   The view transform rotation along the +x axis in radians
 * @param   y   The view transform rotation along the +y axis in radians
 *
 * @return  void
 */
void ViewContext::setRotation( float x, float y )
{
    // adjust for rotations greater than pi rads
    while ( x > ( 2 * M_PI ) ) x -= 2 * M_PI;
    while ( y > ( 2 * M_PI ) ) y -= 2 * M_PI;

    // adjust for rotations less than negative pi rads
    while ( x < 0 ) x += 2 * M_PI;
    while ( y < 0 ) y += 2 * M_PI;

    // set rotations
    viewRotationX = x;
    viewRotationY = y;

    update();
}


/**
 * @brief   Sets the scale of the view context's transformation matrix
 *
 * @param   x   The view transformation scale along the +x axis
 * @param   y   The view transformation scale along the +y axis
 * @param   z   The view transformation scale along the +z axis
 *
 * @return  void
 */
void ViewContext::setScale( float x, float y, float z )
{
    viewScaleX = x;
    viewScaleY = y;
    viewScaleZ = z;
    update();
}


/**
 * @brief   Gets the view translation of the view context's transformation
 *          matrix
 *
 * @param   void
 *
 * @return  The translation of the view context's transformation matrix as a
 *          Vector3
 */
Vector3<float> ViewContext::getTranslation() const
{
    return Vector3<float>(
        viewTranslationX,
        viewTranslationY,
        viewTranslationZ
    );
}


/**
 * @brief   Gets the view rotation of the view context's transformation matrix
 *
 * @param   void
 *
 * @return  The rotation of the view context's transformation matrix as a
 *          Vector3
 */
Vector2<float> ViewContext::getRotation() const
{
    return Vector2<float>(
        viewRotationX,
        viewRotationY
    );
}


/**
 * @brief   Gets the view scale of the view context's transformation matrix
 *
 * @param   void
 *
 * @return  The scale of the view context's transformation matrix as a Vector3
 */
Vector3<float> ViewContext::getScale() const
{
    return Vector3<float>(
        viewScaleX,
        viewScaleY,
        viewScaleZ
    );
}


/**
 * @brief   Resets the view translation of the transformation matrix
 *
 * @param   void
 *
 * @return  void
 */
void ViewContext::resetTranslation()
{
    setTranslation(
        DEFAULT_VIEW_TRANSLATION_X,
        DEFAULT_VIEW_TRANSLATION_Y,
        DEFAULT_VIEW_TRANSLATION_Z
    );
}


/**
 * @brief   Resets the view rotation of the transformation matrix
 *
 * @param   void
 *
 * @return  void
 */
void ViewContext::resetRotation()
{
    setRotation(
        DEFAULT_VIEW_ROTATION_X,
        DEFAULT_VIEW_ROTATION_Y
    );
}


/**
 * @brief   Resets the view scale of the transformation matrix
 *
 * @param   void
 *
 * @return  void
 */
void ViewContext::resetScale()
{
    setScale(
        DEFAULT_VIEW_SCALE_X,
        DEFAULT_VIEW_SCALE_Y,
        DEFAULT_VIEW_SCALE_Z
    );
}


/**
 * @brief   Resets the view transformation matrix
 *
 * @param   void
 *
 * @return  void
 */
void ViewContext::resetView()
{
    resetTranslation();
    resetRotation();
    resetScale();
}


/**
 * @brief   Updates the view context's transformation matrices
 *
 * @param   void
 *
 * @return  void
 */
void ViewContext::update()
{
    // determine transformation matrix
    transform =
            genScreenTranslationMatrix() *
            genScreenFlipMatrix()        *
            genViewScaleMatrix()         *
            genViewRotationMatrix()      *
            genViewTranslationMatrix();

    // determine inverse transformation matrix
    invTransform =
            genInvViewTranslationMatrix()    *
            genInvViewRotationMatrix()       *
            genInvViewScaleMatrix()          *
            genInvScreenFlipMatrix()         *
            genInvScreenTranslationMatrix();
}


/**
 * @brief   Writes the current transformation matrix to the output stream
 *
 * @param   &os     The output stream to write to
 *
 * @return  The output stream
 */
std::ostream &ViewContext::out( std::ostream &os ) const
{
    transform.out( os );
    return os;
}


/* ---------------------------- Private Functions --------------------------- */


/**
 * @brief   Pans the current perspective across the view plane's x-axis
 *
 * @param   magnitude   The magnitude of the pan
 *
 * @return  void
 */
void ViewContext::panX( float magnitude )
{
    // get Y rotation
    float rotationY = getRotation().getY();

    // determine view plane x translation
    int quadrant = ( ( int ) ( rotationY / ( M_PI_2 ) ) ) % 4;
    float axisRotation = rotationY - ( quadrant * M_PI_2 );

    // determine tangent ratio
    float oppAdjRatio = tan( axisRotation );

    // determine axis-translation signs and magnitudes based on quadrant
    float signX;
    float signZ;
    float translateX;
    float translateZ;

    switch ( quadrant )
    {
        case 0:
            signX = 1;
            signZ = 1;
            translateX = 1;
            translateZ = oppAdjRatio;
            break;

        case 1:
            signX = -1;
            signZ = 1;
            translateX = oppAdjRatio;
            translateZ = 1;
            break;

        case 2:
            signX = -1;
            signZ = -1;
            translateX = 1;
            translateZ = oppAdjRatio;
            break;

        case 3:
            signX = 1;
            signZ = -1;
            translateX = oppAdjRatio;
            translateZ = 1;
            break;

        default:
            throw ViewContextException( "Illegal view quadrant." );
    }

    // normalize translation hypotenuse to magnitude
    float thetaX = atan( translateX / translateZ );
    float thetaZ = atan( translateZ / translateX );
    translateX = signX * magnitude * sin( thetaX );
    translateZ = signZ * magnitude * sin( thetaZ );

    // std::cout << "QUAD:  " << quadrant << std::endl;
    // std::cout << "SCALE: " << magnitude << std::endl;
    // std::cout << "TX:    " << translateX << std::endl;
    // std::cout << "TZ:    " << translateZ << std::endl;
    // std::cout << "HYP:   " << sqrt( pow( translateX, 2 ) + pow( translateZ, 2 ) ) << std::endl;
    // std::cout << std::endl;

    // translate
    translate( translateX, 0, translateZ );
}


/**
 * @brief   Pans the current perspective across the view plane's y-axis
 *
 * @param   magnitude   The magnitude of the pan
 *
 * @return  void
 */
void ViewContext::panY( float magnitude )
{
    // get X rotation
    float rotationX = getRotation().getX();

    // determine view plane x translation
    int quadrant = ( ( int ) ( rotationX / ( M_PI_2 ) ) ) % 4;
    float axisRotation = rotationX - ( quadrant * M_PI_2 );

    // determine tangent ratio
    float oppAdjRatio = tan( axisRotation );

    // determine axis-translation signs and magnitudes based on quadrant
    float signY;
    float translateY;
    float translateN;

    switch ( quadrant )
    {
        case 0:
            signY = 1;
            translateY = 1;
            translateN = oppAdjRatio;
            break;

        case 1:
            signY = -1;
            translateY = oppAdjRatio;
            translateN = 1;
            break;

        case 2:
            signY = 1;
            translateY = 1;
            translateN = oppAdjRatio;
            break;

        case 3:
            signY = 1;
            translateY = oppAdjRatio;
            translateN = 1;
            break;

        default:
            throw ViewContextException( "Illegal view quadrant." );
    }

    // normalize translation hypotenuse to magnitude
    float thetaY = atan( translateY / translateN );
    translateY = signY * magnitude * sin( thetaY );

    Point3D lookTarget = getLookVector();

    lookTarget.setX( -1 * magnitude * lookTarget.getX() );
    lookTarget.setY( 0 );
    lookTarget.setZ( -1 * magnitude * lookTarget.getZ() );

    // std::cout << "QUAD:  " << quadrant << std::endl;
    // std::cout << "SCALE: " << magnitude << std::endl;
    // std::cout << "TX:    " << translateY << std::endl;
    // std::cout << "TZ:    " << translateN << std::endl;
    // std::cout << "HYP:   " << sqrt( pow( translateY, 2 ) + pow( translateN, 2 ) ) << std::endl;
    // std::cout << "LOOK:  " << getLookVector() << std::endl;
    // std::cout << std::endl;

    // translate
    translate( lookTarget.getX(), translateY, lookTarget.getZ() );
}


/**
 * @brief   Generates the view translation matrix from internal parameters
 *
 * @param   void
 *
 * @return  The view translation matrix
 */
Matrix<float> ViewContext::genViewTranslationMatrix() const
{
    Matrix<float> viewTranslationMatrix = Matrix<float>( 4, 4 );

    viewTranslationMatrix[0][0] = 1;
    viewTranslationMatrix[1][1] = 1;
    viewTranslationMatrix[2][2] = 1;
    viewTranslationMatrix[3][3] = 1;

    viewTranslationMatrix[0][3] = viewTranslationX;
    viewTranslationMatrix[1][3] = viewTranslationY;
    viewTranslationMatrix[2][3] = viewTranslationZ;

    // std::cout << std::endl << "View Translation: " << std::endl;
    // viewTranslationMatrix.out(std::cout);

    return viewTranslationMatrix;
}


/**
 * @brief   Generates the view rotation matrix from internal parameters
 *
 * @param   void
 *
 * @return  The view rotation matrix
 */
Matrix<float> ViewContext::genViewRotationMatrix() const
{
    // generate view rotation X
    Matrix<float> viewRotationXMatrix = Matrix<float>( 4, 4 );

    viewRotationXMatrix[0][0] = 1;
    viewRotationXMatrix[3][3] = 1;

    viewRotationXMatrix[1][1] = cos( viewRotationX );
    viewRotationXMatrix[1][2] = -sin( viewRotationX );
    viewRotationXMatrix[2][1] = sin( viewRotationX );
    viewRotationXMatrix[2][2] = cos( viewRotationX );

    // generate view rotation Y
    Matrix<float> viewRotationYMatrix = Matrix<float>( 4, 4 );

    viewRotationYMatrix[1][1] = 1;
    viewRotationYMatrix[3][3] = 1;

    viewRotationYMatrix[0][0] = cos( viewRotationY );
    viewRotationYMatrix[2][0] = -sin( viewRotationY );
    viewRotationYMatrix[0][2] = sin( viewRotationY );
    viewRotationYMatrix[2][2] = cos( viewRotationY );

    // generate combined rotation matrix x -> y
    Matrix<float> viewRotationMatrix =
        viewRotationXMatrix * viewRotationYMatrix;

    // std::cout << std::endl << "View Rotation: " << std::endl;
    // viewRotationMatrix.out(std::cout);

    return viewRotationMatrix;
}


/**
 * @brief   Generates the view scale matrix from internal parameters
 *
 * @param   void
 *
 * @return  The view scale matrix
 */
Matrix<float> ViewContext::genViewScaleMatrix() const
{
    Matrix<float> viewScaleMatrix = Matrix<float>( 4, 4 );

    viewScaleMatrix[0][0] = viewScaleX;
    viewScaleMatrix[1][1] = viewScaleY;
    viewScaleMatrix[2][2] = viewScaleZ;
    viewScaleMatrix[3][3] = 1;

    // std::cout << std::endl << "View Scale: " << std::endl;
    // viewScaleMatrix.out(std::cout);

    return viewScaleMatrix;
}


/**
 * @brief   Generates the inverse view translation matrix from internal parameters
 *
 * @param   void
 *
 * @return  The inverse view translation matrix
 */
Matrix<float> ViewContext::genInvViewTranslationMatrix() const
{
    Matrix<float> invViewTranslationMatrix = genViewTranslationMatrix();

    invViewTranslationMatrix[0][3] = -1 * invViewTranslationMatrix[0][3];
    invViewTranslationMatrix[1][3] = -1 * invViewTranslationMatrix[1][3];
    invViewTranslationMatrix[2][3] = -1 * invViewTranslationMatrix[2][3];

    // std::cout << std::endl << "Inverse View Translation: " << std::endl;
    // invViewTranslationMatrix.out(std::cout);

    return invViewTranslationMatrix;
}


/**
 * @brief   Generates the inverse view rotation matrix from internal parameters
 *
 * @param   void
 *
 * @return  The inverse view rotation matrix
 */
Matrix<float> ViewContext::genInvViewRotationMatrix() const
{
    Matrix<float> invViewRotationMatrix = genViewRotationMatrix();

    invViewRotationMatrix = ~invViewRotationMatrix;

    // std::cout << std::endl << "Inverse View Rotation: " << std::endl;
    // invViewRotationMatrix.out(std::cout);

    return invViewRotationMatrix;
}


/**
 * @brief   Generates the inverse view scale matrix from internal parameters
 *
 * @param   void
 *
 * @return  The inverse view scale matrix
 */
Matrix<float> ViewContext::genInvViewScaleMatrix() const
{
    Matrix<float> invViewScaleMatrix = genViewScaleMatrix();

    invViewScaleMatrix[0][0] = 1 / invViewScaleMatrix[0][0];
    invViewScaleMatrix[1][1] = 1 / invViewScaleMatrix[1][1];
    invViewScaleMatrix[2][2] = 1 / invViewScaleMatrix[2][2];

    // std::cout << std::endl << "Inverse View Scale: " << std::endl;
    // invViewScaleMatrix.out(std::cout);

    return invViewScaleMatrix;
}


/**
 * @brief   Generates the screen translation matrix from internal parameters
 *
 * @param   void
 *
 * @return  The screen translation matrix
 */
Matrix<float> ViewContext::genScreenTranslationMatrix() const
{
    Matrix<float> screenTranslationMatrix = Matrix<float>( 4, 4 );

    screenTranslationMatrix[0][0] = 1;
    screenTranslationMatrix[1][1] = 1;
    screenTranslationMatrix[2][2] = 1;
    screenTranslationMatrix[3][3] = 1;

    screenTranslationMatrix[0][3] = ( ( float ) gc->getWindowWidth() ) / 2;
    screenTranslationMatrix[1][3] = ( ( float ) gc->getWindowHeight() ) / 2;

    // std::cout << std::endl << "Screen Translation: " << std::endl;
    // screenTranslationMatrix.out( std::cout);

    return screenTranslationMatrix;
}


/**
 * @brief   Generates the screen flip matrix from internal parameters
 *
 * @param   void
 *
 * @return  The screen flip matrix
 */
Matrix<float> ViewContext::genScreenFlipMatrix() const
{
    Matrix<float> screenFlipMatrix = Matrix<float>( 4, 4 );

    screenFlipMatrix[0][0] = 1;
    screenFlipMatrix[1][1] = -1;
    screenFlipMatrix[2][2] = 1;
    screenFlipMatrix[3][3] = 1;

    // std::cout << std::endl << "Screen Flip: " << std::endl;
    // screenFlipMatrix.out( std::cout);

    return screenFlipMatrix;
}


/**
 * @brief   Generates the inverse screen translation matrix from internal parameters
 *
 * @param   void
 *
 * @return  The inverse screen translation matrix
 */
Matrix<float> ViewContext::genInvScreenTranslationMatrix() const
{
    Matrix<float> invScreenTranslationMatrix = genScreenTranslationMatrix();

    invScreenTranslationMatrix[0][3] = -1 * invScreenTranslationMatrix[0][3];
    invScreenTranslationMatrix[1][3] = -1 * invScreenTranslationMatrix[1][3];
    invScreenTranslationMatrix[2][3] = -1 * invScreenTranslationMatrix[2][3];

    // std::cout << std::endl << "Inverse Screen Translation: " << std::endl;
    // invScreenTranslationMatrix.out( std::cout);

    return invScreenTranslationMatrix;
}


/**
 * @brief   Generates the inverse screen flip matrix from internal parameters
 *
 * @param   void
 *
 * @return  The inverse screen flip matrix
 */
Matrix<float> ViewContext::genInvScreenFlipMatrix() const
{
    Matrix<float> invScreenFlipMatrix = genScreenFlipMatrix();

    invScreenFlipMatrix[0][0] = 1 / invScreenFlipMatrix[0][0];
    invScreenFlipMatrix[1][1] = 1 / invScreenFlipMatrix[1][1];
    invScreenFlipMatrix[2][2] = 1 / invScreenFlipMatrix[2][2];

    // std::cout << std::endl << "Inverse Screen Flip: " << std::endl;
    // invScreenFlipMatrix.out( std::cout);

    return invScreenFlipMatrix;
}


/* -------------------------- Overloaded Operators -------------------------- */


/**
 * @brief   Writes a view context's transformation matrix to the output stream
 *
 * @param   &os     The output stream to write to
 * @param   &vc     The view context to write
 *
 * @return  The output stream
 */
std::ostream &operator<<( std::ostream &os, const ViewContext &vc )
{
    vc.out( os );
    return os;
}


/* -------------------------------------------------------------------------- */