/* --------------------------------- Header --------------------------------- */


/**
 * @file    line.cpp
 * @brief   Line shape
 */


/* -------------------------------- Includes -------------------------------- */


# include <sstream>

# include "line.h"
# include "viewcontext.h"


/* ----------------------- Constructors / Destructors ----------------------- */


/**
 * @brief   Creates a black line from two points
 *
 * @param   &start  The starting point of the line
 * @param   &end    The ending point of the line
 *
 * @return  The created line
 */
Line::Line( const Point3D &start, const Point3D &end ):
Line( start, end, Color( 0, 0, 0 ) )
{}


/**
 * @brief   Creates a colored line from two points
 *
 * @param   &start  The starting point of the line
 * @param   &end    The ending point of the line
 * @param   &color  The color of the line
 *
 * @return  The created line
 */
Line::Line( const Point3D &start, const Point3D &end, const Color &color ):
Shape( color, midpoint( start, end ) ),
verts( Vector2<Point3D*>( start.clone(), end.clone() ) )
{}


/**
 * @brief   Creates a line from an existing line
 *
 * @param   &line   The line to create from
 *
 * @return  The created line
 */
Line::Line( const Line &line ):
Shape( line.color, line.origin ), verts( line.verts )
{}


/**
 * @brief   Line destructor
 *
 * @param   void
 *
 * @return  void
 */
Line::~Line()
{
    Point3D::vector2DeepDelete( verts );
}


/* -------------------------- Overloaded Operators -------------------------- */


/**
 * @brief   Assigns a line to this line
 *
 * @param   &line   The line to assign to this line
 *
 * @return  A reference to this line
 */
Line &Line::operator=( const Line &line )
{
    Point3D::vector2DeepDelete( verts );

    this->verts = Point3D::vector2DeepCopy( line.verts );
    this->color = line.color;
    this->origin = line.origin;

    return *this;
}


/**
 * @brief   Gets an immutable reference to the vertex at the specified index
 *
 * @param   index   The index of the vertex
 *
 * @return  An immutable reference to the vertex at the specified index
 */
const Point3D &Line::operator[]( unsigned int index ) const
{
    return *verts[index];
}


/**
 * @brief   Gets a mutable reference to the vertex at the specified index
 *
 * @param   index   The index of the vertex
 *
 * @return  A mutable reference to the vertex at the specified index
 */
Point3D &Line::operator[]( unsigned int index )
{
    return *verts[index];
}


/**
 * @brief   Converts a line to a string and writes it to an output stream
 *
 * @param   &os     The output stream to write to
 * @param   &line   The line to convert
 *
 * @return  The output stream
 */
std::ostream &operator<<( std::ostream &os, const Line &line )
{
    line.out( os );
    return os;
}


/* ---------------------------- Public Functions ---------------------------- */


/**
 * @brief   Draws this line to an graphics context
 *
 * @param   *gc     The graphics context to draw to
 *
 * @return  void
 */
void Line::draw( GraphicsContext *gc) const
{
    // convert from model to device coordinates
    Point3D start = *verts[0];
    Point3D end = *verts[1];
    // draw line
    gc->drawLine(
        ( int ) start.getX(), ( int ) start.getY(),
        ( int ) end.getX(),   ( int ) end.getY()
    );
}


/**
 * @brief   Clones this line
 *
 * @param   void
 *
 * @return  The cloned copy of this line
 */
Line *Line::clone() const
{
    return new Line( *verts[0], *verts[1], color );
}


/**
 * @brief   Converts this line to a string and flushes it to an output
 *          stream
 *
 * @param   &os     The output stream to write to
 *
 * @return  A reference to the output stream
 */
std::ostream &Line::out( std::ostream &os ) const
{
    Shape::out( os );
    os << "  VERTICES( " << *verts[0] << " " << *verts[1] << " )";
    return os;
}


/* ---------------------------- Private Functions --------------------------- */


/**
 * @brief   Determines the midpoint between two points
 *
 * @param   &a   The first point
 * @param   &b   The second point
 *
 * @return  The midpoint between two points
 */
Point3D Line::midpoint( const Point3D &a, const Point3D &b )
{
    return Point3D(
        ( a.getX() + b.getX() ) / 2,
        ( a.getY() + b.getY() ) / 2,
        ( a.getZ() + b.getZ() ) / 2 );
}


/* -------------------------------------------------------------------------- */