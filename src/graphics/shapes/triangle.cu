/* --------------------------------- Header --------------------------------- */


/**
 * @file    triangle.cpp
 * @brief   Triangle shape
 */


/* -------------------------------- Includes -------------------------------- */


# include <sstream>
# include "color.h"
# include "triangle.h"


/* --------------------- Constructors / Destructors --------------------- */


/**
 * @brief   Creates a black triangle from three points
 *
 * @param   &start  The starting point of the triangle
 * @param   &mid    The middle point of the triangle
 * @param   &end    The ending point of the triangle
 *
 * @return  The created triangle
 */
Triangle::Triangle( const Point3D &start, const Point3D &mid, const Point3D &end ):
Triangle( start, mid, end, Color( 0, 0, 0 ) )
{}


/**
 * @brief   Creates a colored triangle from three points
 *
 * @param   &start  The starting point of the triangle
 * @param   &mid    The middle point of the triangle
 * @param   &end    The ending point of the triangle
 * @param   &color  The color of the triangle
 *
 * @return  The created triangle
 */
Triangle::Triangle( const Point3D &start, const Point3D &mid, const Point3D &end, const Color &color ):
Shape( color, midpoint( start, mid, end ) ),
verts( Vector3<Point3D*>( start.clone(), mid.clone(), end.clone() ) )
{}


/**
 * @brief   Creates a triangle from an existing triangle
 *
 * @param   &triangle   The triangle to create from
 *
 * @return  The created triangle
 */
Triangle::Triangle( const Triangle &triangle ) :
Shape( triangle.color, triangle.origin ), verts( triangle.verts )
{}


/**
 * @brief   Triangle destructor
 *
 * @param   void
 *
 * @return  void
 */
Triangle::~Triangle()
{
    Point3D::vector3DeepDelete( verts );
}


/* -------------------------- Overloaded Operators -------------------------- */


/**
 * @brief   Assigns a triangle to this triangle
 *
 * @param   &triangle   The triangle to assign to this triangle
 *
 * @return  A reference to this triangle
 */
Triangle &Triangle::operator=( const Triangle &triangle )
{
    Point3D::vector3DeepDelete( verts );

    this->verts = Point3D::vector3DeepCopy( triangle.verts );
    this->color = triangle.color;
    this->origin = triangle.origin;

    return *this;
}


/**
 * @brief   Gets an immutable reference to the vertex at the specified index
 *
 * @param   index   The index of the vertex
 *
 * @return  An immutable reference to the vertex at the specified index
 */
const Point3D &Triangle::operator[]( unsigned int index ) const
{
    return *verts[index];
}


/**
 * @brief   Gets a mutable reference to the vertex at the specified index
 *
 * @param   index   The index of the vertex
 *
 * @return  A mutable reference to the vertex at the specified index
 */
Point3D &Triangle::operator[]( unsigned int index )
{
    return *verts[index];
}


/**
 * @brief   Converts a triangle to a string and writes it to an output stream
 *
 * @param   &os         The output stream to write to
 * @param   &triangle   The triangle to convert
 *
 * @return  The output stream
 */
std::ostream &operator<<( std::ostream &os, const Triangle &triangle )
{
    triangle.out( os );
    return os;
}


/* ---------------------------- Public Functions ---------------------------- */


/**
 * @brief   Draws this shape to a graphics context
 *
 * @param   *gc     The graphics context to draw to
 *
 * @return  void
 */
void Triangle::draw( GraphicsContext *gc ) const
{
    // convert from model to device coordinates
    Point3D start = *verts[0];
    Point3D mid = *verts[1];
    Point3D end = *verts[2];

    // draw lines
    gc->drawLine( ( int ) start.getX(), ( int ) start.getY(),
                  ( int ) mid.getX(),   ( int ) mid.getY() );

    gc->drawLine( ( int ) mid.getX(),   ( int ) mid.getY(),
                  ( int ) end.getX(),   ( int ) end.getY() );

    gc->drawLine( ( int ) end.getX(),   ( int ) end.getY(),
                  ( int ) start.getX(), ( int ) start.getY() );
}


/**
 * @brief   Clones this triangle
 *
 * @param   void
 *
 * @return  The cloned copy of this triangle
 */
Triangle *Triangle::clone() const
{
    return new Triangle( *verts[0], *verts[1], *verts[2], color );
}


/**
 * @brief   Converts this triangle to a string and flushes it to an output
 *          stream
 *
 * @param   &os     The output stream to write to
 *
 * @return  A reference to the output stream
 */
std::ostream &Triangle::out( std::ostream &os ) const
{
    Shape::out( os );
    os << "  VERTICES( " << *verts[0] << " "
                         << *verts[1] << " "
                         << *verts[2] << " )";
    return os;
}


/* ---------------------------- Private Functions --------------------------- */

/**
 * @brief   Determines the midpoint between three points
 *
 * @param   &a  The first point
 * @param   &b  The second point
 * @param   &c  The third point
 *
 * @return  The midpoint between three points
 */
Point3D Triangle::midpoint( const Point3D &a, const Point3D &b, const Point3D &c )
{
    return Point3D(
        ( a.getX() + b.getX() + c.getX() ) / 3,
        ( a.getY() + b.getY() + c.getY() ) / 3,
        ( a.getZ() + b.getZ() + c.getZ() ) / 3 );
}

/* -------------------------------------------------------------------------- */