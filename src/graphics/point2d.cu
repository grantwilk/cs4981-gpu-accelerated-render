#include "hip/hip_runtime.h"
/* --------------------------------- Header --------------------------------- */


/**
 * @file    point2d.cpp
 * @brief   2D point class
 */


/* -------------------------------- Includes -------------------------------- */


# include <algorithm>
# include <cmath>

# include "point2d.h"


/* ----------------------- Constructors / Destructors ----------------------- */


/**
 * @brief   Creates a 2D point at coordinates (0, 0)
 *
 * @param   x   The x-coordinate
 * @param   y   The y-coordinate
 *
 * @return  The created point
 */
Point2D::Point2D():
Point2D( 0, 0 )
{}


/**
 * @brief   Creates a 2D point at the specified coordinates
 *
 * @param   x   The x-coordinate
 * @param   y   The y-coordinate
 *
 * @return  The created point
 */
Point2D::Point2D( float x, float y ):
Vector2<float>( x, y )
{}


/**
 * @brief   Creates a 2D point from a 2D vector of floats
 *
 * @param   &v  The vector to create the 2D point from
 *
 * @return  The created point
 */
Point2D::Point2D( const Vector2<float> &v ):
Point2D( v[0], v[1] )
{}


/**
 * @brief   Creates a 2D point from a 2x1 matrix of floats
 *
 * @param   &m  The matrix to create the 2D point from
 *
 * @return  The created point
 */
Point2D::Point2D( const Matrix<float> &m ):
Point2D( m[0][0], m[1][0] )
{
    if ( ( m.getRows() != 2 ) || ( m.getColumns() != 1 ) )
    {
        throw MatrixException( "Point2D construction matrix size mismatch" );
    }
}


/**
 * @brief   Creates a 2D point from an existing 2D point
 *
 * @param   point   The 2D point to create from
 *
 * @return  The created point
 */
Point2D::Point2D( const Point2D &p ):
Point2D( p.getX(), p.getY() )
{}


Point2D::~Point2D() = default;


/* -------------------------- Overloaded Operators -------------------------- */


/**
 * @brief   Assigns a point to this point
 *
 * @param   &point  The point to assign
 *
 * @return  A reference to this point
 */
Point2D &Point2D::operator=( const Point2D &p )
{
    setX( p.getX() );
    setY( p.getY() );
    return *this;
}


/**
 * @brief   Adds a 2D point to this 2D point and assigns the sum to
 *          this 2D point
 *
 * @param   &point  The 2D point to add
 *
 * @return  A reference to this 2D point
 */
Point2D &Point2D::operator+=( const Point2D &p )
{
    return *this = *this + p;
}


/**
 * @brief   Subtracts a 2D point from this 2D point and assigns the
 *          difference to this 2D point
 *
 * @param   &point  The 2D point to subtract
 *
 * @return  A reference to this 2D point
 */
Point2D &Point2D::operator-=( const Point2D &p )
{
    return *this = *this - p;
}


/**
 * @brief   Adds a 2D point to this 2D point
 *
 * @param   &point  The 2D point to add
 *
 * @return  The sum of the 2D points
 */
Point2D Point2D::operator+( const Point2D &p ) const
{
    return Point2D( getX() + p.getX(), getY() + p.getY() );
}


/**
 * @brief   Subtracts a 2D point from this 2D point
 *
 * @param   &point  The 2D point to subtract
 *
 * @return  The difference of the 2D points
 */
Point2D Point2D::operator-( const Point2D &p ) const
{
    return Point2D( getX() - p.getX(), getY() - p.getY() );
}


/**
 * @brief   Determines if the magnitude of this point is less than the
 *          magnitude of another point
 *
 * @param   &p  The point to compare against
 *
 * @return  True if the magnitude of this point is less than the magnitude
 *          of the other point, false otherwise
 */
bool Point2D::operator<( const Point2D &p ) const
{
    return magnitude() < p.magnitude();
}


/**
 * @brief   Determines if the magnitude of this point is less than or equal
 *          to the magnitude of another point
 *
 * @param   &p  The point to compare against
 *
 * @return  True if the magnitude of this point is less than or equal to
 *          the magnitude of the other point, false otherwise
 */
bool Point2D::operator<=( const Point2D &p ) const
{
    return magnitude() <= p.magnitude();
}


/**
 * @brief   Determines if the magnitude of this point is greater than the
 *          magnitude of another point
 *
 * @param   &p  The point to compare against
 *
 * @return  True if the magnitude of this point is greater than the magnitude
 *          of the other point, false otherwise
 */
bool Point2D::operator>( const Point2D &p ) const
{
    return magnitude() > p.magnitude();
}


/**
 * @brief   Determines if the magnitude of this point is greater than or
 *          equal to the magnitude of another point
 *
 * @param   &p  The point to compare against
 *
 * @return  True if the magnitude of this point is greater than or equal to
 *          the magnitude of the other point, false otherwise
 */
bool Point2D::operator>=( const Point2D &p ) const
{
    return magnitude() >= p.magnitude();
}


/**
 * @brief   Converts a 2D point to a string and writes it to an output stream
 *
 * @param   &os     The output stream to write to
 * @param   &point  The 2D point to convert
 *
 * @return  The output stream
 */
std::ostream &operator<<( std::ostream &os, const Point2D &p )
{
    p.out( os );
    return os;
}


/* ---------------------------- Public Functions ---------------------------- */



/**
 * @brief   Creates a dynamically allocated clone of this 2D point
 *
 * @param   void
 *
 * @return  A pointer to cloned 2D point
 */
Point2D *Point2D::clone() const
{
    return new Point2D( getX(), getY() );
}


/**
 * @brief   Calculates a transformed copy of this 2D point
 *
 * @param   &m  The 2D transformation matrix
 *
 * @return  A transformed copy of this 2D point
 */
Point2D Point2D::transform( const Matrix<float> &m ) const
{
    if ( ( m.getRows() != 3 ) || ( m.getColumns() != 3 ) )
    {
        throw MatrixException( "Point2D transformation matrix size mismatch." );
    }

    Matrix<float> transformVector = m * Vector3<float>( getX(), getY(), 1 );

    return Point2D( transformVector[0][0], transformVector[1][0] );
}


/**
 * @brief   Gets the magnitude of the vector between the origin and this
 *          2D point
 *
 * @param   void
 *
 * @return  The magnitude of the vector between the origin and this 2D point
 */
float Point2D::magnitude() const
{
    return sqrt( pow( getX(), 2 ) + pow( getY(), 2 ) );
}


/**
 * @brief   Gets the magnitude of the angle between the origin-vector formed by
 *          this 2D point and the origin-vector formed by another 2D point in
 *          radians
 *
 * @param   void
 *
 * @return  The angle of the origin-vector formed by this 2D point and the
 *          origin-vector formed by another 2D point in radians
 */
float Point2D::angle( const Point2D &p ) const
{
    float dot = this->dot( p );
    float det = getX() * p.getY() - getY() * p.getX();
    return atan2( det, dot );
}


/**
 * @brief   Converts this 2D point to a string and writes it to an output
 *          stream
 *
 * @param   &os     The output stream to write to
 *
 * @return  The output stream
 */
std::ostream &Point2D::out( std::ostream &os ) const
{
    os << "POINT2D( " << getX() << " " << getY() << " )";
    return os;
}


/**
 * @brief   Performs a deep copy of a vector of 2D point pointers
 *
 * @param   &v      The vector to copy
 *
 * @return  The copied vector
 */
std::vector<Point2D*> Point2D::vectorDeepCopy( const std::vector<Point2D*> &v )
{
    std::vector<Point2D*> clone = std::vector<Point2D*>();

    std::for_each( v.begin(), v.end(), [&clone]( Point2D *p )
                   {
                       Point2D *pClone = p->clone();
                       clone.push_back( pClone );
                   }
    );

    return clone;
}


/**
 * @brief   Performs a deep delete of a vector of 2D point pointers
 *
 * @param   &v      The vector to delete
 *
 * @return  void
 */
void Point2D::vectorDeepDelete( const std::vector<Point2D*> &v )
{
    std::for_each( v.begin(), v.end(), []( Point2D *p )
                   {
                       delete p;
                   }
    );
}


/**
 * @brief   Performs a deep copy a 2D vector of 2D point pointers
 *
 * @param   &v      The vector to copy
 *
 * @return  The copied vector
 */
Vector2<Point2D*> Point2D::vector2DeepCopy( const Vector2<Point2D*> &v )
{
    return Vector2<Point2D*>( v[0]->clone(), v[1]->clone() );
}


/**
 * @brief   Performs a deep deletion of a 2D vector of 2D point pointers
 *
 * @param   &v      The vector to delete
 *
 * @return  void
 */
void Point2D::vector2DeepDelete( const Vector2<Point2D*> &v )
{
    delete v.getX();
    delete v.getY();
}


/**
 * @brief   Performs a deep copy a 3D vector of 2D point pointers
 *
 * @param   &v      The vector to copy
 *
 * @return  The copied vector
 */
Vector3<Point2D*> Point2D::vector3DeepCopy( const Vector3<Point2D*> &v )
{
    return Vector3<Point2D*>( v[0]->clone(), v[1]->clone(), v[2]->clone() );
}


/**
 * @brief   Performs a deep deletion of a 3D vector of 2D point pointers
 *
 * @param   &v      The vector to delete
 *
 * @return  void
 */
void Point2D::vector3DeepDelete( const Vector3<Point2D*> &v )
{
    delete v.getX();
    delete v.getY();
    delete v.getZ();
}


/**
 * @brief   Performs a deep copy a 4D vector of 2D point pointers
 *
 * @param   &v      The vector to copy
 *
 * @return  The copied vector
 */
Vector4<Point2D*> Point2D::vector4DeepCopy( const Vector4<Point2D*> &v )
{
    return Vector4<Point2D*>( v[0]->clone(), v[1]->clone(), v[2]->clone(), v[3]->clone() );
}


/**
 * @brief   Performs a deep deletion of a 3D vector of 2D point pointers
 *
 * @param   &v      The vector to delete
 *
 * @return  void
 */
void Point2D::vector4DeepDelete( const Vector4<Point2D*> &v )
{
    delete v.getX();
    delete v.getY();
    delete v.getZ();
    delete v.getW();
}


/* -------------------------------------------------------------------------- */