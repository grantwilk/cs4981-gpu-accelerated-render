#include "hip/hip_runtime.h"
/* --------------------------------- Header --------------------------------- */


/**
 * @file    shapecontainer.cpp
 * @brief   Set container of shape pointers
 */


/* -------------------------------- Includes -------------------------------- */


# include <algorithm>
# include <set>
# include <sstream>
#include <hip/driver_types.h>

# include "cudaerr.cuh"
# include "shape.h"
# include "shapecontainer.h"
# include "triangle.h"


/* ----------------------- Constructors / Destructors ----------------------- */


const unsigned int SHAPE_DIM = 3;
const unsigned int VERT_DIM = 4;


/* ----------------------- Constructors / Destructors ----------------------- */


/**
 * @brief   Creates an empty shape container
 *
 * @param   void
 *
 * @return  The created shape container
 */
ShapeContainer::ShapeContainer() = default;


/**
 * @brief   Creates a shape container from an existing shape container
 *
 * @param   &sc     The shape container to create from
 *
 * @return  The created shape container
 */
ShapeContainer::ShapeContainer( const ShapeContainer &sc )
{
    add( sc );
}


/**
 * @brief   Shape container destructor
 *
 * @param   void
 *
 * @return  void
 */
ShapeContainer::~ShapeContainer()
{
    std::for_each( shapes.begin(), shapes.end(), [this]( Shape *shape )
       {
           delete shape;
       }
    );
}


/* -------------------------- Overloaded Operators -------------------------- */


/**
 * @brief   Assigns the shapes from another shape container to this
 *          shape container
 *
 * @param   &sc     The shape container to assign from
 *
 * @return  This shape container
 */
ShapeContainer &ShapeContainer::operator=( const ShapeContainer &sc )
{
    erase();
    add( sc );
    return *this;
}


/**
 * @brief   Converts a shape container to a string and writes it to an
 *          output stream
 *
 * @param   &os     The output stream to write to
 * @param   &sc     The shape container to convert
 *
 * @return  The output stream
 */
std::ostream &operator<<( std::ostream &os, const ShapeContainer &sc )
{
    sc.out( os );
    return os;
}


/* ---------------------------- Public Functions ---------------------------- */


/**
 * Pushes this shape container to the GPU device
 */
void ShapeContainer::pushToDevice()
{
    // input shapes buffer
    float inputShapes[shapes.size() * SHAPE_DIM * VERT_DIM];

    // copy each tri to input memory
    for (unsigned int shapeIdx = 0; shapeIdx < shapes.size(); shapeIdx++)
    {
        unsigned int shapeOffset = shapeIdx * SHAPE_DIM * VERT_DIM;
        for (unsigned int vertIdx = 0; vertIdx < SHAPE_DIM; vertIdx++)
        {
            unsigned int vertOffset = vertIdx * VERT_DIM;
            for (unsigned int coordIdx = 0; coordIdx < VERT_DIM; coordIdx++)
            {
                if (coordIdx < VERT_DIM - 1)
                {
                    inputShapes[shapeOffset + vertOffset + coordIdx] =
                        (*(shapes[shapeIdx]))[vertIdx][coordIdx];
                }
                else
                {
                    inputShapes[shapeOffset + vertOffset + coordIdx] = 1;
                }
            }
        }
    }

    // free existing device mallocs if they exist
    if ( d_inputShapes != nullptr)
    {
        HANDLE_CUDA_ERROR(hipFree(d_inputShapes));
    }
    if ( d_outputShapes != nullptr)
    {
        HANDLE_CUDA_ERROR(hipFree(d_outputShapes));
    }

    // malloc new input and output
    HANDLE_CUDA_ERROR(
        hipMalloc(&d_inputShapes, shapes.size() * SHAPE_DIM * VERT_DIM * sizeof(float))
    );
    HANDLE_CUDA_ERROR(
        hipMalloc(&d_outputShapes, shapes.size() * SHAPE_DIM * VERT_DIM * sizeof(float))
    );

    // copy shapes to device
    HANDLE_CUDA_ERROR(
        hipMemcpy(
            (void *) d_inputShapes,
            (void *) inputShapes,
            shapes.size() * SHAPE_DIM * VERT_DIM * sizeof(float),
            hipMemcpyHostToDevice
        )
    );
}


/**
 * @brief   Adds a shape to this shape container
 *
 * @param   &shape  The shape to add
 *
 * @return  void
 */
void ShapeContainer::add( const Shape &shape )
{
    shapes.insert( shapes.end(), shape.clone() );
}


/**
 * @brief   Adds the elements from another shape container to this
 *          shape container
 *
 * @param   &sc  The shape container to add
 *
 * @return  void
 */
void ShapeContainer::add( const ShapeContainer &sc )
{
    std::for_each( sc.shapes.begin(), sc.shapes.end(), [this]( Shape *shape )
       {
           add( *shape );
       }
    );
}


/**
 * @brief   Draws the shapes in this shape container
 *
 * @param   *gc     The graphics context to draw to
 * @param   *vc     The view context to draw with
 *
 * @return  void
 */
void ShapeContainer::draw( GraphicsContext *gc, ViewContext *vc ) const
{
    // input shapes buffer
    float outputShapes[shapes.size() * SHAPE_DIM * VERT_DIM];

    // copy view transform to local
    float viewTransform[VERT_DIM][VERT_DIM];

    for (unsigned int row = 0; row < VERT_DIM; row++)
    {
        for (unsigned int col = 0; col < VERT_DIM; col++)
        {
            viewTransform[row][col] = ViewContext::transform[row][col];
        }
    }

    // copy view transform to device
    HANDLE_CUDA_ERROR(
        hipMemcpy(
            ( void * ) ViewContext::d_viewTransform,
            ( void * ) viewTransform,
            VERT_DIM * VERT_DIM * sizeof( float ),
            hipMemcpyHostToDevice
        )
    );

    // zero output matrix
    HANDLE_CUDA_ERROR(
        hipMemset(d_outputShapes, 0, shapes.size() * SHAPE_DIM * VERT_DIM * sizeof(float))
    );

    // run GPU kernel
    unsigned int blocks = ceil(shapes.size() / 1024.0);

    applyViewTransform<<<blocks, 1024>>>(
        d_inputShapes,
        d_outputShapes,
        ViewContext::d_viewTransform
    );
    HANDLE_CUDA_ERROR(hipDeviceSynchronize());

    // copy shapes from device
    HANDLE_CUDA_ERROR(
        hipMemcpy(
            (void *) outputShapes,
            (void *) d_outputShapes,
            shapes.size() * SHAPE_DIM * VERT_DIM * sizeof(float),
            hipMemcpyDeviceToHost
        )
    );

    // parse output points into output shapes vector
    std::vector<Shape*> parsedShapes;

    for (unsigned int shapeIdx = 0; shapeIdx < shapes.size(); shapeIdx++)
    {
        unsigned int shapeOffset = shapeIdx * SHAPE_DIM * VERT_DIM;

        Point3D verts[3];

        for (unsigned int vertIdx = 0; vertIdx < SHAPE_DIM; vertIdx++)
        {
            unsigned int vertOffset = vertIdx * VERT_DIM;

            verts[vertIdx].setX(outputShapes[shapeOffset + vertOffset + 0]);
            verts[vertIdx].setY(outputShapes[shapeOffset + vertOffset + 1]);
            verts[vertIdx].setZ(outputShapes[shapeOffset + vertOffset + 2]);
        }

        Triangle * tri = new Triangle(verts[0], verts[1], verts[2]);
        parsedShapes.insert(parsedShapes.end(), tri);
    }

    // draw shapes
    std::for_each(parsedShapes.begin(), parsedShapes.end(), [gc](Shape *shape)
       {
           shape->draw(gc);
       }
    );
}


/**
 * @brief   Converts the shapes in this shape container to strings and
 *          outputs them to an output stream
 *
 * @param   &os     The output stream to write to
 *
 * @return  The output stream
 */
std::ostream &ShapeContainer::out( std::ostream &os ) const
{
    std::for_each( shapes.begin(), shapes.end(), [&os]( Shape *shape )
       {
           shape->out( const_cast<std::ostream&>( os ) );
           os << std::endl;
       }
    );
    return os;
}


/**
 * @brief   Removes all shapes from this shape container
 *
 * @param   void
 *
 * @return  void
 */
void ShapeContainer::erase()
{
    std::for_each( shapes.begin(), shapes.end(), []( Shape *shape )
       {
           delete shape;
       }
    );
    shapes.clear();
}


/**
 * @brief   Gets the size of the shape container
 *
 * @param   void
 *
 * @return  The size of the shape container
 */
unsigned int ShapeContainer::size()
{
    return shapes.size();
}


/* ------------------------------ GPU Kernels ------------------------------- */


__global__ void applyViewTransform(
    float * inputShapes, float * outputShapes, float * viewTransform
)
{
    unsigned int shapeIdx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int shapeOffset = shapeIdx * SHAPE_DIM * VERT_DIM;

    // matrix vector multiplication
    for (unsigned int vertIdx = 0; vertIdx < SHAPE_DIM; vertIdx++)
    {
        unsigned int vertOffset = vertIdx * VERT_DIM;

        for (unsigned int row = 0; row < VERT_DIM; row++)
        {

            unsigned int rowOffset = row * VERT_DIM;

            for (unsigned int i = 0; i < VERT_DIM; i++)
            {
                outputShapes[shapeOffset + vertOffset + row] +=
                    viewTransform[rowOffset + i] * inputShapes[shapeOffset + vertOffset + i];
            }
        }
    }
}


/* -------------------------------------------------------------------------- */